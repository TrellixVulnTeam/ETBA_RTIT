#include "hip/hip_runtime.h"
#include "buffer_copy.cuh"
#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <math.h>

#define CHECK_CUDA(cond) check_cuda(cond, __LINE__)

void check_cuda(hipError_t status, std::size_t line)
{
    if(status != hipSuccess)
    {
        std::cout << hipGetErrorString(status) << '\n';
        std::cout << "Line: " << line << '\n';
        throw 0;
    }
}

__global__ void copy_kernel(float* output, const float* input, int N, const int* stepOverList, int singleVol)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // for (int i = blockIdx.x * blockDim.x + threadIdx.x;  i < N; i += blockDim.x * gridDim.x) {

        // int map_i = stepOverList[i/singleVol]*singleVol+i%singleVol;

        // output[i] = input[i];
        // int cur_num = int(i/singleVol);
        // int stored_num = 0;
        // if (indicator[cur_num] == 1) {
        //     for (int j = 0; j < cur_num; j++) {
        //         stored_num += indicator[j];
        //     }
        //     output[i % singleVol + singleVol * stored_num] = input[i];
        // }
    if (i < N) {
        int map_i = stepOverList[i/singleVol]*singleVol+i%singleVol;
        output[i] = input[map_i];
    }
}

void buffercopy(float* d_vector_dest, const float* d_vector_src, int sz, const int* stepOverList, int singleVol, const hipStream_t& stream)
{
    // int grid_size = 0, block_size = 0;
    // CHECK_CUDA(hipOccupancyMaxPotentialBlockSize(&grid_size, &block_size, copy_kernel, 0));
    // std::cout << "Grid size: " << grid_size << "  Block size: " << block_size << std::endl;

    copy_kernel<<<singleVol, sz/singleVol, 0, stream>>>(d_vector_dest, d_vector_src, sz, stepOverList, singleVol);
    CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void foo()
{
    printf("CUDA!\n");
}

void useCUDA()
{
    foo<<<1,5>>>();
    CHECK_CUDA(hipDeviceSynchronize());
}